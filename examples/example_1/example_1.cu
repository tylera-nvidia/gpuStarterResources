
#include <hip/hip_runtime.h>
#include <iostream>

#include <nvtx3/nvToolsExt.h>

#define IDIVUP(a, b) (((a) + (b) - 1) / (b))

////////////////////////////////////////////////////////////////////////////////
///
///
///
///
////////////////////////////////////////////////////////////////////////////////
template< typename T > 
__launch_bounds__(1024, 2)
__global__ void customElementMultiple(T *pMatA, T *pMatB, T *pMatC, uint32_t matSizeX, uint32_t matSizeY)
{
  int numElements = matSizeX * matSizeY;
  int gridThreads = gridDim.x * blockDim.x;
  
  for(int curIdx = threadIdx.x + blockIdx.x * blockDim.x; curIdx < numElements; curIdx+=gridThreads )
  {
    pMatC[curIdx] = pMatA[curIdx] * pMatB[curIdx];
  }
  
}


////////////////////////////////////////////////////////////////////////////////
///
///
///
///
////////////////////////////////////////////////////////////////////////////////
template< typename T > 
void manual_cuda( int numSamples = 25)
{
  nvtxRangePushA("Manual CUDA Version");

  // initialize timing variables
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int warmupOffset = 5; //run a few times to warm up kernels  
  
  uint32_t matSizeX = 4096;
  uint32_t matSizeY = 4096;
  uint32_t dataSize =  matSizeX * matSizeY * sizeof(T);
  
  
  T *pMatA;
  T *pMatB;
  T *pMatC;
  
  hipMalloc(&pMatA, dataSize);
  hipMalloc(&pMatB, dataSize);
  hipMalloc(&pMatC, dataSize);
  
  int blockSize =  1024; //hardcoded as largest block size
  
  // int numBlocks = IDIVUP(matSizeX, blockSize);
  
  // simple scale out
  // int numBlocks = IDIVUP(matSizeX, blockSize) * matSizeY;
  // scale out but not as much
  // int numBlocks = IDIVUP(matSizeX, blockSize) * matSizeY/100;
  
  //programatic
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int numBlocks = deviceProp.multiProcessorCount * 2; 
  
  
  #pragma unroll
  for( int curSample = 0; curSample < numSamples; curSample++)
  {
    nvtxRangePushA("Iteration");
    if(curSample == warmupOffset )
    {
      //start of timing
      hipEventRecord(start, 0);      
    }    
    
    customElementMultiple<T><<<numBlocks,blockSize>>>(pMatA, pMatB, pMatC, matSizeX, matSizeY);
    
    nvtxRangePop();
  }
  
  //end of timing
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  
   //report average time
  float time_ms;
  hipEventElapsedTime(&time_ms, start, stop);
  std::cout << "Average elapsed time per iteration is: " << time_ms * 1.0e3 / static_cast<double>(numSamples - warmupOffset)  << "us" << std::endl;
 
  nvtxRangePop();
}

////////////////////////////////////////////////////////////////////////////////
///
///
///
///
////////////////////////////////////////////////////////////////////////////////
int main()
{
  nvtxRangePushA("main");
  manual_cuda<float>();
  nvtxRangePop();
}

